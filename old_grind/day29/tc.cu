#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h> // for tensor cores



#define WARP_SIZE 4 // to execute threads in groups of warps, and each warp consists of 4 threads

// MMA matrix tile dimensions
#define M 16
#define N 16
#define K 16

// GEMM configuration.
#define M_TILES 256
#define N_TILES 256
#define K_TILES 256

#define M_TOTAL (M * M_TILES)
#define N_TOTAL (N * N_TILES)
#define K_TOTAL (K * K_TILES)

// the above define -> M,N,K & M TILES, N TILES, K TILES get together to make a matrix of size 4096 * 4096

using namespace nvcuda;

// init matrix
__host__ void InitMatrix(half *A, half *B, float *C) {
    for (int i = 0; i < M_TOTAL * K_TOTAL; i++)
        A[i] = __float2half(rand() % 1000 / 1000.0f);
    for (int i = 0; i < K_TOTAL * N_TOTAL; i++)
        B[i] = __float2half(rand() % 1000 / 1000.0f);
    for (int i = 0; i < M_TOTAL * N_TOTAL; i++)
        C[i] = rand() % 1000 / 1000.0f;
}

// wmma(warp-synchronous matrix multiply-accumulate) kernel for fp16
__global__ void WMMAF16TensorCore(half *A, half *B, float *C, float *D) {
    int ix = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // declares small tile-sized matrices to fit into Tensor Cores
    // initializes the accumulator to zero before multiplication
    // prepares for efficient mma_sync operations using Tensor Cores
    wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, M, N, K, float> ab_frag;
    wmma::fragment<wmma::accumulator, M, N, K, float> c_frag;

    wmma::fill_fragment(ab_frag, 0.0f);

    // AB = A * B
    int a_row = ix * M;
    int b_row = iy * N;
    for (int k = 0; k < K_TOTAL; k += K) {
        int a_col = k;
        int b_col = k;

        if (a_row < M_TOTAL && a_col < K_TOTAL && b_row < K_TOTAL && b_col < N_TOTAL) {
            // load the inputs
            wmma::load_matrix_sync(a_frag, A + a_col + a_row * K_TOTAL, K_TOTAL);
            wmma::load_matrix_sync(b_frag, B + b_col + b_row * N_TOTAL, N_TOTAL);

            // perform the matrix multiplication
            wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
        }
    }

    // D = AB + C
    int c_row = a_row;
    int c_col = b_row;
    if (c_row < M_TOTAL && c_col < N_TOTAL) {
        wmma::load_matrix_sync(c_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);

        for (int i = 0; i < c_frag.num_elements; i++) {
            c_frag.x[i] = ab_frag.x[i] + c_frag.x[i];
        }

        // store the output
        wmma::store_matrix_sync(D + c_col + c_row * N_TOTAL, c_frag, N_TOTAL, wmma::mem_row_major);
    }
}

hipError_t CalcWMMA(half *A, half *B, float *C, float *D) {
    hipError_t cuda_status;
    dim3 gridDim, blockDim;

    // 16 warps in one block
    blockDim.x = 4 * WARP_SIZE;
    blockDim.y = 4;

    gridDim.x = (M_TOTAL + (M * blockDim.x / WARP_SIZE - 1)) / (M * blockDim.x / WARP_SIZE);
    gridDim.y = (N_TOTAL + N * blockDim.y - 1) / (N * blockDim.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    WMMAF16TensorCore<<<gridDim, blockDim>>>(A, B, C, D);
    cuda_status = hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);

    printf("[+] GPU (with Tensor Cores) Elapsed Time: %f ms\n", time);
    printf("[+] TFLOPS: %.2f\n", ((double)M_TOTAL * N_TOTAL * K_TOTAL * 2) / (time * 1e9));

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return cuda_status;
}

int main() {
    hipError_t cuda_status;
    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        printf("hipSetDevice failed! ");
        return 1;
    }

    // matrix on device
    half *A;
    half *B;
    float *C;
    float *D;

    // CUDA Unified Memory
    hipMallocManaged((void **)&A, sizeof(half) * M_TOTAL * K_TOTAL);
    hipMallocManaged((void **)&B, sizeof(half) * K_TOTAL * N_TOTAL);
    hipMallocManaged((void **)&C, sizeof(float) * M_TOTAL * N_TOTAL);
    hipMallocManaged((void **)&D, sizeof(float) * M_TOTAL * N_TOTAL);

    // initialize matrices
    printf("[*] Initializing Matrix...\n");
    InitMatrix(A, B, C);
    printf("[+]   A: %d x %d\n", M_TOTAL, K_TOTAL);
    printf("[+]   B: %d x %d\n", K_TOTAL, N_TOTAL);
    printf("[+]   C: %d x %d\n", M_TOTAL, N_TOTAL);

    // compute D = A * B + C using Tensor Cores
    printf("[*] Computing D = A * B + C with Tensor Cores...\n");
    cuda_status = CalcWMMA(A, B, C, D);

    if (cuda_status != hipSuccess) {
        printf("Kernel execution failed! ");
        return 1;
    }

    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(D);

    cuda_status = hipDeviceReset();
    if (cuda_status != hipSuccess) {
        printf("hipDeviceReset failed! ");
        return 1;
    }

    return 0;
}
// [+] GPU (with Tensor Cores) Elapsed Time: 43.589630 ms
// [+] TFLOPS: 3.15